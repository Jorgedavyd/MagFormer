#include "hip/hip_runtime.h"
// implementation of the cross transformer

/*The general idea is to paralellize the forward method of both heads
with kernelized implementations of grouped query self attention with a 
residual layer all in one.
The cross transformer will have three kernels that will be used in the forward pass:
self attention

cross attention

ffn

All of them already with the residual layer included.

We'll create 2 streams in which each head will be working.

This will kernelize every aspect of the cross transformer, allowing for faster training.

The backward pass as well.
*/

#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <torch/extension.h>

// Defining the grouped query attention kernel
__global__ void GroupedQueryAttentionKernel (hipdnnTensorDescriptor_t * queries, hipdnnTensorDescriptor_t * keys, hipdnnTensorDescriptor_t * values);

// Defining the Multi query attention kernel
__global__ void MultiQueryAttentionKernel (hipdnnTensorDescriptor_t * queries, hipdnnTensorDescriptor_t * key, hipdnnTensorDescriptor_t * value);

// Defining the Multi head attention kernel
__global__ void MultiHeadAttention (hipdnnTensorDescriptor_t * queries, hipdnnTensorDescriptor_t * keys, hipdnnTensorDescriptor_t * values);

// # Defining a general attention module
// parameters: n_q, n_kv (int) (per_group query and (key and values) number of elements per group)
// Each block will be designed to take care of a different group

template <typename scalar_t>
__global__ void GeneralSelfAttentionKernel (
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> queries,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> keys,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> values,
    torch::PackedTensorAccessor32<scalar_t, 2, torch::RestrictPtrTraits> global_out,
    float[][] * Wq, float[][] * Wk, float[][] * Wv, float[][] * W_fc,
    const unsigned int groups, const unsigned int n_queries, const unsigned int q_dim, 
    const unsigned int k_dim, const unsigned int v_dim, const unsigned int batch_size, const unsigned int seq_len,
    const float scale_factor, const unsigned int shared_query_size, const unsigned int shared_kv_size
) {
    /*
    Shared memory:
    shared_query -> (seq, q_dim) -> (groups, n_queries, s, head_dim)
    shared_key -> (seq, k_dim) -> (groups, s, head_dim)
    shared_value -> (seq, v_dim) -> (groups, s, head_dim)

    assertion: groups*n_queries*s*head_dim >= seq*q_dim
    
    */
    extern __shared__ float sharedMemory[];
    
    float* shared_query = sharedMemory;
    float* shared_key = shared_query + sequence_length * groups * head_dim;
    float* shared_value = shared_key + sequence_length * groups * head_dim;
    float* shared_out = shared_value + sequence_length * groups * head_dim * n_queries;

    const unsigned int b_x = blockIdx.x;
    const unsigned int x = threadIdx.x + b_x * blockDim.x;
    const unsigned int y = threadIdx.y + b_x * blockDim.x;
    const unsigned int z = threadIdx.z + b_x * blockDim.x;

    
    //Here, b_x works as a batch dimension, z makes the decision to choose between the query, key, value.
    
    if (b_x < batch_size && z < 3) {
        switch z {
            case 0: 
                shared_query[seq_len * y + x] = queries[b_x][y][x].item<float>();
                __syncthreads();
            case 1: 
                shared_key[seq_len * y + x] = keys[b_x][y][x].item<float>(); 
                __syncthreads();
            case 2: 
                shared_value[seq_len * y + x] = values[b_x][y][x].item<float>(); 
                __syncthreads();
    }
    }

    // Synchronize for if and switch declarations
    __syncthreads();

    // compute the forward method of the weights
    

    // Make the computation of the attention

    // Compute the softmax
    
    // Put the value into global memory
    global_out[b_x][y][x] = out;

}

void ASSERTIONS (
    torch::Tensor * queries, torch::Tensor * keys, torch::Tensor * values,
    torch::Tensor * weight_q, torch::Tensor * weight_k, torch::Tensor * weight_v,
    torch::Tensor * weight_fc, const unsigned int groups, const unsigned int head_dim
) {
    /*
    # Checks assertions for general attention.


    */
    
}

torch::Tensor GeneralAttention (torch::Tensor * queries, torch::Tensor * keys, torch::Tensor * values, 
                                    torch::Tensor * weight_q, torch::Tensor * weight_k, torch::Tensor * weight_v, 
                                    torch::Tensor * weight_fc, const unsigned int groups, const unsigned int head_dim) {
    // Check assertions
    ASSERTIONS(queries, keys, values, weight_q, weight_k, weight_v, weight_fc, groups, head_dim)

    // Defining parameters for linear transformation
    const unsigned int q_dim, k_dim, v_dim, batch_size, sequence_length; 
    
    batch_size = queries.size(0);
    sequence_length = queries.size(1);

    /*
    1. Queries: batch, seq, q_dim
    2. Keys: batch, seq, k_dim
    3. values: batch, seq, v_dim
    */

    q_dim = queries.size(-1);
    k_dim = keys.size(-1);
    v_dim = values.size(-1);

    // Defining the tensor that will be allocated in global memory waiting for the kernel execution.
    torch::Tensor out = torch::empty({batch_size, sequence_length, q_dim});
    
    // Defining the weights and biases into constant memory.
    __constant__ float Wq[q_dim][groups * n_queries * head_dim];
    __constant__ float Wk[k_dim][groups * head_dim];
    __constant__ float Wv[v_dim][groups * head_dim];
    __constant__ float W_fc[groups * n_queries * head_dim][groups * n_queries * head_dim];

    // Sending the weights from global memory to constant memory.
    hipMemcpyToSymbol(HIP_SYMBOL(Wq), weight_q.data_ptr<float>(), q_dim * groups * n_queries * head_dim * size(float));
    hipMemcpyToSymbol(HIP_SYMBOL(Wk), weight_k.data_ptr<float>(), k_dim * groups * head_dim * size(float));
    hipMemcpyToSymbol(HIP_SYMBOL(Wv), weight_v.data_ptr<float>(), v_dim * groups * head_dim * size(float));
    hipMemcpyToSymbol(HIP_SYMBOL(W_fc), weight_fc.data_ptr<float>(), (groups * n_queries * head_dim)^2 * size(float));

    /*
    # Kernel general overview:
    ## Resources:
    
    register:
        float ...

    shared_memory:
        shared_query[s * n_queries * head_dim], shared_key[s * head_dim], shared_value[s * head_dim], shared_out[s * n_queries * head_dim]  
    
    constant_memory:
        Wq[], Wk[], Wv[], W_fc[]
    
    global_memory:
        torch::Tensor queries, keys, values, out;

    ## Routine:
    1. Define parameters.
    2. Define the weights into the constant memory.
    kernel:
        3. Define the query, key, value into the shared memory.
        4. Make the compute -> recompute for the shared_out.
        5. Compute the softmax and multiply by the computed value.
        6. Send to the global memory.
    3. Return the tensor.
    */

    // Launching kernel
    AT_DISPATCH_FLOATING_TYPES(queries.scalar_type(), "GeneralAttentionKernel", ([&] {
        GeneralAtttentionKernel<scalar_t><<<blocksPerGrid, threadsPerBlock>>>(
            queries.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            keys.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            values.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            out.packed_accessor32<scalar_t, 2, torch::RestrictPtrTraits>(),
            &Wq, &Wk, &Wv, &W_fc, groups, n_queries, q_dim, k_dim, v_dim
            );

    return out;

}



